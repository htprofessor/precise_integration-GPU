#include <iostream>
#include <fstream>
#include <sstream>
#include <vector>
#include <chrono>
#include <hip/hip_runtime.h>
#include <hipblas.h>

// 定义处理512阶矩阵的常量
#define MATRIX_SIZE 512

// CUDA 错误检查宏
#define CUDA_CHECK(call)                                                    \
do {                                                                        \
    hipError_t err = call;                                                 \
    if (err != hipSuccess) {                                               \
        std::cerr << "CUDA error in " << __FILE__ << " at line " << __LINE__; \
        std::cerr << ": " << hipGetErrorString(err) << std::endl;          \
        exit(1);                                                            \
    }                                                                       \
} while (0)

// CuBLAS 错误检查宏
#define CUBLAS_CHECK(call)                                                  \
do {                                                                        \
    hipblasStatus_t status = call;                                           \
    if (status != HIPBLAS_STATUS_SUCCESS) {                                  \
        std::cerr << "CuBLAS error in " << __FILE__ << " at line " << __LINE__; \
        std::cerr << ": " << status << std::endl;                           \
        exit(1);                                                            \
    }                                                                       \
} while (0)

void matrix_exponential(const double* d_A, double* d_expAt, int n, double delt, int N, hipblasHandle_t handle);
bool read_matrix_from_file(const std::string& filename, std::vector<double>& matrix, int& n);
void matrix_multiply(const double* d_A, const double* d_B, double* d_C, int n, hipblasHandle_t handle);
void matrix_add(double* d_A, const double* d_B, int n, hipblasHandle_t handle);
void matrix_square(const double* d_A, double* d_C, int n, hipblasHandle_t handle);

int main() {
    // 记录程序开始时间
    auto program_start = std::chrono::high_resolution_clock::now();

    // 读取512阶矩阵文件
    std::string matrixAFile = "matrixA_512.txt";
    std::vector<double> A;
    int nA;

    if (!read_matrix_from_file(matrixAFile, A, nA)) {
        std::cerr << "读取矩阵时出错" << std::endl;
        return -1;
    }

    if (nA != MATRIX_SIZE) {
        std::cerr << "矩阵的维度不是512阶" << std::endl;
        return -1;
    }

    int n = nA;

    // 动态分配GPU内存
    double *d_A, *d_expAt;
    CUDA_CHECK(hipMalloc(&d_A, n * n * sizeof(double)));
    CUDA_CHECK(hipMalloc(&d_expAt, n * n * sizeof(double)));

    // 将矩阵 A 从主机传输到设备
    CUDA_CHECK(hipMemcpy(d_A, A.data(), n * n * sizeof(double), hipMemcpyHostToDevice));

    // 创建CuBLAS句柄
    hipblasHandle_t handle;
    CUBLAS_CHECK(hipblasCreate(&handle));

    // 时间步长和迭代次数
    double delt = 0.0000001;  // 时间步长
    int N = 20;               // 精细积分法迭代次数

    // 记录开始时间
    auto start = std::chrono::high_resolution_clock::now();

    // 计算矩阵指数
    matrix_exponential(d_A, d_expAt, n, delt, N, handle);

    // 记录结束时间
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed = end - start;

    // 将结果从设备传回主机
    std::vector<double> expAt(n * n);
    CUDA_CHECK(hipMemcpy(expAt.data(), d_expAt, n * n * sizeof(double), hipMemcpyDeviceToHost));

    // 输出求解结果和运行时间
    std::cout << "矩阵指数计算时间: " << elapsed.count() << " 秒" << std::endl;

    // 输出部分结果以验证正确性
    std::cout << "exp(A * delt) 的前4x4块：\n";
    for (int i = 0; i < 4; ++i) {
        for (int j = 0; j < 4; ++j) {
            std::cout << expAt[i * n + j] << " ";
        }
        std::cout << std::endl;
    }

    // 释放设备内存
    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_expAt));

    // 销毁CuBLAS句柄
    CUBLAS_CHECK(hipblasDestroy(handle));

    auto program_end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> program_elapsed = program_end - program_start;
    std::cout << "程序总运行时间: " << program_elapsed.count() << " 秒" << std::endl;

    return 0;
}

// 实现矩阵指数的计算，使用精细积分法
void matrix_exponential(const double* d_A, double* d_expAt, int n, double delt, int N, hipblasHandle_t handle) {
    double *d_T_a, *d_temp, *d_I;
    CUDA_CHECK(hipMalloc(&d_T_a, n * n * sizeof(double)));
    CUDA_CHECK(hipMalloc(&d_temp, n * n * sizeof(double)));
    CUDA_CHECK(hipMalloc(&d_I, n * n * sizeof(double)));

    // 初始化 T_a = A * delt * (I + (A * delt) / 2)
    const double alpha = delt;
    CUBLAS_CHECK(hipblasDcopy(handle, n * n, d_A, 1, d_T_a, 1));  // T_a = A
    CUBLAS_CHECK(hipblasDscal(handle, n * n, &alpha, d_T_a, 1));   // T_a = A * delt

    // temp = A * delt / 2
    CUBLAS_CHECK(hipblasDcopy(handle, n * n, d_T_a, 1, d_temp, 1));  // temp = A * delt
    const double beta = 0.5;
    CUBLAS_CHECK(hipblasDscal(handle, n * n, &beta, d_temp, 1));     // temp = A * delt / 2

    // T_a = A * delt * (I + (A * delt) / 2)
    const double one = 1.0;
    CUBLAS_CHECK(hipblasDaxpy(handle, n * n, &one, d_temp, 1, d_T_a, 1));  // T_a = A * delt + (A * delt / 2)

    // 递推计算 T_a
    for (int iter = 0; iter < N; iter++) {
        // temp = T_a^2
        matrix_square(d_T_a, d_temp, n, handle);

        // T_a = 2 * T_a + T_a^2
        const double two = 2.0;
        CUBLAS_CHECK(hipblasDscal(handle, n * n, &two, d_T_a, 1));  // T_a = 2 * T_a
        CUBLAS_CHECK(hipblasDaxpy(handle, n * n, &one, d_temp, 1, d_T_a, 1));
    }

    // 计算 expAt = I + T_a
    CUBLAS_CHECK(hipblasDcopy(handle, n * n, d_T_a, 1, d_expAt, 1));  // expAt = T_a

    // 构造单位矩阵 I
    std::vector<double> I(n * n, 0.0);
    for (int i = 0; i < n; i++) {
        I[i * n + i] = 1.0;
    }
    CUDA_CHECK(hipMemcpy(d_I, I.data(), n * n * sizeof(double), hipMemcpyHostToDevice));

    // expAt = I + T_a
    matrix_add(d_expAt, d_I, n, handle);

    // 释放临时设备内存
    CUDA_CHECK(hipFree(d_T_a));
    CUDA_CHECK(hipFree(d_temp));
    CUDA_CHECK(hipFree(d_I));
}

// 从文件读取矩阵
bool read_matrix_from_file(const std::string& filename, std::vector<double>& matrix, int& n) {
    std::ifstream file(filename);
    if (!file.is_open()) {
        std::cerr << "无法打开文件: " << filename << std::endl;
        return false;
    }

    std::string line;
    std::vector<double> temp_matrix;
    int row_count = 0;

    while (std::getline(file, line)) {
        std::istringstream iss(line);
        double value;
        int col_count = 0;
        while (iss >> value) {
            temp_matrix.push_back(value);
            col_count++;
        }
        if (row_count == 0) {
            n = col_count;  // 确定矩阵的维度
        } else if (col_count != n) {
            std::cerr << "文件格式不正确，列数不匹配" << std::endl;
            return false;
        }
        row_count++;
    }

    matrix = std::move(temp_matrix);  // 将临时矩阵赋值给目标矩阵
    return true;
}

void matrix_multiply(const double* d_A, const double* d_B, double* d_C, int n, hipblasHandle_t handle) {
    const double alpha = 1.0, beta = 0.0;
    CUBLAS_CHECK(hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, &alpha, d_A, n, d_B, n, &beta, d_C, n));
}

void matrix_add(double* d_A, const double* d_B, int n, hipblasHandle_t handle) {
    const double alpha = 1.0;
    CUBLAS_CHECK(hipblasDaxpy(handle, n * n, &alpha, d_B, 1, d_A, 1));
}

void matrix_square(const double* d_A, double* d_C, int n, hipblasHandle_t handle) {
    matrix_multiply(d_A, d_A, d_C, n, handle);
}
